#include "hip/hip_runtime.h"

#include "../include/somigliana/green_core.h"
#include "../include/somigliana/green_core.inl"

#include <Eigen/Dense>

__global__ void mvc_kernel(scalar_t *d_PHI,
                           const scalar_t *d_V,
                           const index_t  *d_cageF,
                           const scalar_t *d_cageV,
                           const index_t nv,
                           const index_t ncf,
                           const index_t ncv) {
  unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;

  mvc_kernel(d_PHI,
    d_V,
    d_cageF,
    d_cageV,
    nv,
    ncf,
    ncv,
    index);
}

__global__ void green_kernel(scalar_t *d_phix,
                             scalar_t *d_phiy,
                             scalar_t *d_phiz,
                             scalar_t *d_psi,
                             const scalar_t *d_V,
                             const index_t  *d_cageF,
                             const scalar_t *d_cageV,
                             const scalar_t *d_cageN,
                             const index_t nv,
                             const index_t ncf,
                             const index_t ncv,
                             const scalar_t *d_qp,
                             const scalar_t *d_qw,
                             const index_t nq) {
  unsigned int thread_index = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int index = thread_index/ncf, f = thread_index%ncf;

  green_kernel(d_phix,
    d_phiy,
    d_phiz,
    d_psi,
    d_V,
    d_cageF,
    d_cageV,
    d_cageN,
    nv,
    ncf,
    ncv,
    d_qp,
    d_qw,
    nq,
    index,
    f);
}

__global__ void green_kernel_post(scalar_t *d_phi,
                                  const scalar_t *d_phix,
                                  const scalar_t *d_phiy,
                                  const scalar_t *d_phiz,
                                  const index_t *d_cageF,
                                  const index_t nv,
                                  const index_t ncf,
                                  const index_t ncv) {
  unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;

  green_kernel_post(d_phi,
    d_phix,
    d_phiy,
    d_phiz,
    d_cageF,
    nv,
    ncf,
    ncv,
    index);
}

__global__ void somig_kernel(const scalar_t nu,
                             scalar_t *d_PHIx,
                             scalar_t *d_PHIy,
                             scalar_t *d_PHIz,
                             scalar_t *d_PSI ,
                             const scalar_t *d_V,
                             const index_t  *d_cageF,
                             const scalar_t *d_cageV,
                             const scalar_t *d_cageN,
                             const index_t nv,
                             const index_t ncf,
                             const index_t ncv,
                             const scalar_t *d_qp,
                             const scalar_t *d_qw,
                             const index_t nq)  {
  unsigned int thread_index = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int index = thread_index/ncf,
                f = thread_index%ncf;

  somig_kernel(nu,
    d_PHIx,
    d_PHIy,
    d_PHIz,
    d_PSI,
    d_V,
    d_cageF,
    d_cageV,
    d_cageN,
    nv,
    ncf,
    ncv,
    d_qp,
    d_qw,
    nq,
    index,
    f);
  }

__global__ void somig_kernel_post(scalar_t *d_PHI,
                                  const scalar_t *d_PHIx,
                                  const scalar_t *d_PHIy,
                                  const scalar_t *d_PHIz,
                                  const index_t *d_cageF,
                                  const index_t nv,
                                  const index_t ncf) {
  unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;

  somig_kernel_post(d_PHI,
    d_PHIx,
    d_PHIy,
    d_PHIz,
    d_cageF,
    nv,
    ncf,
    index);
}

extern "C" {

  void mvc_gpu(scalar_t *d_PHI,
               const scalar_t *d_V,
               const index_t  *d_cageF,
               const scalar_t *d_cageV,
               const index_t nv,
               const index_t ncf,
               const index_t ncv) {
    const unsigned int blocksize = 256;
    const unsigned int numBlocks = (nv+blocksize-1)/blocksize;
    mvc_kernel<<< numBlocks, blocksize >>>
        (d_PHI, d_V, d_cageF, d_cageV, nv, ncf, ncv);
  }

  void green_gpu(scalar_t *d_phix,
                 scalar_t *d_phiy,
                 scalar_t *d_phiz,
                 scalar_t *d_psi,
                 const scalar_t *d_V,
                 const index_t  *d_cageF,
                 const scalar_t *d_cageV,
                 const scalar_t *d_cageN,
                 const index_t nv,
                 const index_t ncf,
                 const index_t ncv,
                 const scalar_t *d_qp,
                 const scalar_t *d_qw,
                 const index_t nq) {
    // parallel through basis columns
    const unsigned int blocksize = 256;
    const unsigned int numBlocks = (ncf*nv+blocksize-1)/blocksize;
    green_kernel<<< numBlocks, blocksize >>>
        (d_phix, d_phiy, d_phiz, d_psi,
         d_V, d_cageF, d_cageV, d_cageN, nv, ncf, ncv,
         d_qp, d_qw, nq);
  }

  void somig_gpu(const scalar_t nu,
                 scalar_t *d_PHIx,
                 scalar_t *d_PHIy,
                 scalar_t *d_PHIz,
                 scalar_t *d_PSI ,
                 const scalar_t *d_V,
                 const index_t  *d_cageF,
                 const scalar_t *d_cageV,
                 const scalar_t *d_cageN,
                 const index_t nv,
                 const index_t ncf,
                 const index_t ncv,
                 const scalar_t *d_qp,
                 const scalar_t *d_qw,
                 const index_t nq) {
    // parallel through basis entries
    const unsigned int blocksize = 256;
    const unsigned int numBlocks = (ncf*nv+blocksize-1)/blocksize;
    somig_kernel<<< numBlocks, blocksize >>>
        (nu,
         d_PHIx,
         d_PHIy,
         d_PHIz,
         d_PSI,
         d_V,
         d_cageF,
         d_cageV,
         d_cageN,
         nv,
         ncf,
         ncv,
         d_qp,
         d_qw,
         nq);
  }

  // reduce phixyz to phi
  void green_post_gpu(scalar_t *d_phi,
                      const scalar_t *d_phix,
                      const scalar_t *d_phiy,
                      const scalar_t *d_phiz,
                      const index_t  *d_cageF,
                      const index_t nv,
                      const index_t ncf,
                      const index_t ncv) {
    // parallel through basis columns
    const unsigned int blocksize = 256;
    const unsigned int numBlocks = (nv+blocksize-1)/blocksize;
    green_kernel_post<<< numBlocks, blocksize >>>
        (d_phi, d_phix, d_phiy, d_phiz, d_cageF, nv, ncf, ncv);
  }

  // reduce PHIxyz to PHI
  void somig_post_gpu(scalar_t *d_PHI,
                      const scalar_t *d_PHIx,
                      const scalar_t *d_PHIy,
                      const scalar_t *d_PHIz,
                      const index_t  *d_cageF,
                      const index_t nv,
                      const index_t ncf) {
    // parallel through basis columns
    const unsigned int blocksize = 256;
    const unsigned int numBlocks = (nv+blocksize-1)/blocksize;
    somig_kernel_post<<< numBlocks, blocksize >>>
        (d_PHI, d_PHIx, d_PHIy, d_PHIz, d_cageF, nv, ncf);
  }
  
}
